#include "hip/hip_runtime.h"
//
// Created by Michael Staneker on 23.02.21.
//

#include "../include/InitDistribution.cuh"

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) getchar();
    }
}

InitDistribution::InitDistribution(const SimulationParameters p) {

    parameters = p;
    step = 0;
    numParticles = NUM_BODIES;
    numNodes = 3 * numParticles + 12000; //2 * numParticles + 12000;

    // allocate host data
    h_min_x = new float;
    h_max_x = new float;
    h_min_y = new float;
    h_max_y = new float;
    h_min_y = new float;
    h_max_y = new float;

    h_mass = new float[numNodes];

    h_x = new float[numNodes];
    h_y = new float[numNodes];
    h_z = new float[numNodes];

    h_vx = new float[numNodes];
    h_vy = new float[numNodes];
    h_vz = new float[numNodes];

    h_ax = new float[numNodes];
    h_ay = new float[numNodes];
    h_az = new float[numNodes];

    h_child = new int[8*numNodes];
    
    h_start = new int[numNodes];
    h_sorted = new int[numNodes];
    h_count = new int[numNodes];
    h_output = new float[2*numNodes];

    // allocate device data
    gpuErrorcheck(hipMalloc((void**)&d_min_x, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_x, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_min_y, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_y, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_min_z, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_z, sizeof(float)));

    gpuErrorcheck(hipMemset(d_min_x, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_x, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_min_y, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_y, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_min_z, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_z, 0, sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_mass, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_x, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_y, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_z, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_vx, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_vy, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_vz, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_ax, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_ay, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_az, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_index, sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_child, 8*numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_start, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_sorted, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_count, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_mutex, sizeof(int)));

    gpuErrorcheck(hipMemset(d_start, -1, numNodes*sizeof(int)));
    gpuErrorcheck(hipMemset(d_sorted, 0, numNodes*sizeof(int)));

    int memSize = sizeof(float) * 2 * numParticles;

    gpuErrorcheck(hipMalloc((void**)&d_output, 2*numNodes*sizeof(float)));

    plummerModel(h_mass, h_x, h_y, h_z, h_vx, h_vy, h_vz, h_ax, h_ay, h_az, numParticles);

    // copy data to GPU device
    //Changed 2*numParticles*sizeof(float) -> 3*numParticles*sizeof(float)
    hipMemcpy(d_mass, h_mass, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z, h_z, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vx, h_vx, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vy, h_vy, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vz, h_vz, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ax, h_ax, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ay, h_ay, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_az, h_az, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);

}

InitDistribution::~InitDistribution() {
    delete h_min_x;
    delete h_max_x;
    delete h_min_y;
    delete h_max_y;
    delete h_min_z;
    delete h_max_z;

    delete [] h_mass;

    delete [] h_x;
    delete [] h_y;
    delete [] h_z;

    delete [] h_vx;
    delete [] h_vy;
    delete [] h_vz;

    delete [] h_ax;
    delete [] h_ay;
    delete [] h_az;

    delete [] h_child;
    delete [] h_start;
    delete [] h_sorted;
    delete [] h_count;
    delete [] h_output;

    gpuErrorcheck(hipFree(d_min_x));
    gpuErrorcheck(hipFree(d_max_x));
    gpuErrorcheck(hipFree(d_min_y));
    gpuErrorcheck(hipFree(d_max_y));
    gpuErrorcheck(hipFree(d_min_z));
    gpuErrorcheck(hipFree(d_max_z));

    gpuErrorcheck(hipFree(d_mass));

    gpuErrorcheck(hipFree(d_x));
    gpuErrorcheck(hipFree(d_y));
    gpuErrorcheck(hipFree(d_z));

    gpuErrorcheck(hipFree(d_vx));
    gpuErrorcheck(hipFree(d_vy));
    gpuErrorcheck(hipFree(d_vz));

    gpuErrorcheck(hipFree(d_ax));
    gpuErrorcheck(hipFree(d_ay));
    gpuErrorcheck(hipFree(d_az));

    gpuErrorcheck(hipFree(d_index));
    gpuErrorcheck(hipFree(d_child));
    gpuErrorcheck(hipFree(d_start));
    gpuErrorcheck(hipFree(d_sorted));
    gpuErrorcheck(hipFree(d_count));

    gpuErrorcheck(hipFree(d_mutex));

    gpuErrorcheck(hipFree(d_output));

    hipDeviceSynchronize();
}

void InitDistribution::update()
{

    bool timeKernels = false;
    float elapsedTime;
    hipEventCreate(&start_global);
    hipEventCreate(&stop_global);
    hipEventRecord(start_global, 0);

    float elapsedTimeKernel;

    elapsedTimeKernel = kernel::resetArrays(d_mutex, d_x, d_y, d_z, d_mass, d_count, d_start, d_sorted, d_child, d_index,
                        d_min_x, d_max_x, d_min_y, d_max_y, d_min_z, d_max_z, numParticles, numNodes);

    if (timeKernels) {
        std::cout << "\tElapsed time: " << elapsedTimeKernel << " ms" << std::endl;
    }

    elapsedTimeKernel = kernel::computeBoundingBox(d_mutex, d_x, d_y, d_z, d_min_x, d_max_x, d_min_y, d_max_y,
                               d_min_z, d_max_z, numParticles);

    if (timeKernels) {
        std::cout << "\tElapsed time: " << elapsedTimeKernel << " ms"  << std::endl;
    }

    elapsedTimeKernel = kernel::buildTree(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
                      d_min_z, d_max_z, numParticles, numNodes);

    if (timeKernels) {
        std::cout << "\tElapsed time: " << elapsedTimeKernel << " ms"  << std::endl;
    }

    elapsedTimeKernel = kernel::centreOfMass(d_x, d_y, d_z, d_mass, d_index, numParticles);

    if (timeKernels) {
        std::cout << "\tElapsed time: " << elapsedTimeKernel << " ms"  << std::endl;
    }

    elapsedTimeKernel = kernel::sort(d_count, d_start, d_sorted, d_child, d_index, numParticles);

    if (timeKernels) {
        std::cout << "\tElapsed time: " << elapsedTimeKernel << " ms" << std::endl;
    }

    elapsedTimeKernel = kernel::computeForces(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_ax, d_ay, d_az, d_mass, d_sorted, d_child,
                          d_min_x, d_max_x, numParticles, parameters.gravity);

    if (timeKernels) {
        std::cout << "\tElapsed time: " << elapsedTimeKernel << " ms" << std::endl;
    }

    elapsedTimeKernel = kernel::update(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_ax, d_ay, d_az, numParticles,
                   parameters.timestep, parameters.dampening);

    if (timeKernels) {
        std::cout << "\tElapsed time: " << elapsedTimeKernel << " ms" << std::endl;
    }

    if (timeKernels) {
        std::cout << "\tElapsed time: " << elapsedTimeKernel << " ms" << std::endl;
    }

    hipMemcpy(h_x, d_x, 2*numParticles*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_y, d_y, 2*numParticles*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_z, d_z, 2*numParticles*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_vx, d_vx, 2*numParticles*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_vy, d_vy, 2*numParticles*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_vz, d_vz, 2*numParticles*sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    if (timeKernels) {
        std::cout << "\tElapsed time: " << elapsedTimeKernel << " ms" << std::endl;
    }

    std::cout << "x[0]: " << h_x[0] << std::endl;
    std::cout << "v[0]: " << h_vx[0] << std::endl;


    hipEventRecord(stop_global, 0);
    hipEventSynchronize(stop_global);
    hipEventElapsedTime(&elapsedTime, start_global, stop_global);
    hipEventDestroy(start_global);
    hipEventDestroy(stop_global);

    std::cout << "Elapsed time for step " << step << " : " << elapsedTime << " ms" << std::endl;

    step++;
}


void InitDistribution::plummerModel(float *mass, float *x, float* y, float *z,
                                    float *x_vel, float *y_vel, float *z_vel,
                                    float *x_acc, float *y_acc, float *z_acc, int n)
{
    float a = 1.0;
    float pi = 3.14159265;
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(0, 1.0);
    std::uniform_real_distribution<float> distribution2(0, 0.1);
    std::uniform_real_distribution<float> distribution_phi(0.0, 2 * pi);
    std::uniform_real_distribution<float> distribution_theta(-1.0, 1.0);

    float check_x_min = 1000;
    float check_x_max = -1000;
    float check_y_min = 1000;
    float check_y_max = -1000;
    float check_z_min = 1000;
    float check_z_max = -1000;

    // loop through all particles
    for (int i = 0; i < n; i++){
        float phi = distribution_phi(generator);
        float theta = acos(distribution_theta(generator));
        float r = a / sqrt(pow(distribution(generator), -0.666666) - 1);

        // set mass and position of particle
        mass[i] = 1.0;
        x[i] = r*cos(phi);
        y[i] = r*sin(phi);
        if (i%2==0) {
            z[i] = i*0.001;
        }
        else {
            z[i] = i*-0.001;
        }

        if (x[i] < check_x_min) {
            check_x_min = x[i];
        }
        if (x[i] > check_x_max) {
            check_x_max = x[i];
        }
        if (y[i] < check_y_min) {
            check_y_min = y[i];
        }
        if (y[i] > check_y_max) {
            check_y_max = y[i];
        }
        if (z[i] < check_z_min) {
            check_z_min = z[i];
        }
        if (z[i] > check_z_max) {
            check_z_max = z[i];
        }

        // set velocity of particle
        float s = 0.0;
        float t = 0.1;
        while(t > s*s*pow(1.0 - s*s, 3.5)){
            s = distribution(generator);
            t = distribution2(generator);
        }
        float v = 100*s*sqrt(2)*pow(1.0 + r*r, -0.25);
        phi = distribution_phi(generator);
        theta = acos(distribution_theta(generator));
        x_vel[i] = v*cos(phi);
        y_vel[i] = v*sin(phi);
        z_vel[i] = 0.0;

        // set acceleration to zero
        x_acc[i] = 0.0;
        y_acc[i] = 0.0;
        z_acc[i] = 0.0;

        if (i%100==0) {
            std::cout << i << ": (" << x[i] << ", " << y[i] << ", " << z[i] << ")"<<std::endl;
        }
    }

    std::cout << "x_max: " << check_x_max << std::endl;
    std::cout << "x_min: " << check_x_min << std::endl;
    std::cout << "y_max: " << check_y_max << std::endl;
    std::cout << "y_min: " << check_y_min << std::endl;
    std::cout << "z_max: " << check_z_max << std::endl;
    std::cout << "z_min: " << check_z_min << std::endl;
}



