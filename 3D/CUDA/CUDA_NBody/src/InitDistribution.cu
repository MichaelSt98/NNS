#include "hip/hip_runtime.h"
//
// Created by Michael Staneker on 23.02.21.
//

#include "../include/InitDistribution.cuh"

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) getchar();
    }
}

InitDistribution::InitDistribution(const SimulationParameters p) {

    parameters = p;
    step = 0;
    numParticles = NUM_BODIES;
    numNodes = 10 * numParticles + 12000; //2 * numParticles + 12000;

    // allocate host data
    h_min_x = new float;
    h_max_x = new float;
    h_min_y = new float;
    h_max_y = new float;
    h_min_y = new float;
    h_max_y = new float;

    h_mass = new float[numNodes];

    h_x = new float[numNodes];
    h_y = new float[numNodes];
    h_z = new float[numNodes];

    h_vx = new float[numNodes];
    h_vy = new float[numNodes];
    h_vz = new float[numNodes];

    h_ax = new float[numNodes];
    h_ay = new float[numNodes];
    h_az = new float[numNodes];

    h_child = new int[8*numNodes];
    
    h_start = new int[numNodes];
    h_sorted = new int[numNodes];
    h_count = new int[numNodes];
    h_output = new float[2*numNodes];

    // allocate device data
    gpuErrorcheck(hipMalloc((void**)&d_min_x, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_x, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_min_y, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_y, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_min_z, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_z, sizeof(float)));

    gpuErrorcheck(hipMemset(d_min_x, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_x, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_min_y, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_y, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_min_z, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_z, 0, sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_mass, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_x, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_y, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_z, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_vx, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_vy, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_vz, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_ax, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_ay, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_az, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_index, sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_child, 8*numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_start, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_sorted, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_count, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_mutex, sizeof(int)));

    gpuErrorcheck(hipMemset(d_start, -1, numNodes*sizeof(int)));
    gpuErrorcheck(hipMemset(d_sorted, 0, numNodes*sizeof(int)));

    int memSize = sizeof(float) * 2 * numParticles;

    gpuErrorcheck(hipMalloc((void**)&d_output, 2*numNodes*sizeof(float)));

    plummerModel(h_mass, h_x, h_y, h_z, h_vx, h_vy, h_vz, h_ax, h_ay, h_az, numParticles);

    // copy data to GPU device
    hipMemcpy(d_mass, h_mass, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z, h_z, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vx, h_vx, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vy, h_vy, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vz, h_vz, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ax, h_ax, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ay, h_ay, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_az, h_az, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);

}

InitDistribution::~InitDistribution() {
    delete h_min_x;
    delete h_max_x;
    delete h_min_y;
    delete h_max_y;
    delete h_min_z;
    delete h_max_z;

    delete [] h_mass;

    delete [] h_x;
    delete [] h_y;
    delete [] h_z;

    delete [] h_vx;
    delete [] h_vy;
    delete [] h_vz;

    delete [] h_ax;
    delete [] h_ay;
    delete [] h_az;

    delete [] h_child;
    delete [] h_start;
    delete [] h_sorted;
    delete [] h_count;
    delete [] h_output;

    gpuErrorcheck(hipFree(d_min_x));
    gpuErrorcheck(hipFree(d_max_x));
    gpuErrorcheck(hipFree(d_min_y));
    gpuErrorcheck(hipFree(d_max_y));
    gpuErrorcheck(hipFree(d_min_z));
    gpuErrorcheck(hipFree(d_max_z));

    gpuErrorcheck(hipFree(d_mass));

    gpuErrorcheck(hipFree(d_x));
    gpuErrorcheck(hipFree(d_y));
    gpuErrorcheck(hipFree(d_z));

    gpuErrorcheck(hipFree(d_vx));
    gpuErrorcheck(hipFree(d_vy));
    gpuErrorcheck(hipFree(d_vz));

    gpuErrorcheck(hipFree(d_ax));
    gpuErrorcheck(hipFree(d_ay));
    gpuErrorcheck(hipFree(d_az));

    gpuErrorcheck(hipFree(d_index));
    gpuErrorcheck(hipFree(d_child));
    gpuErrorcheck(hipFree(d_start));
    gpuErrorcheck(hipFree(d_sorted));
    gpuErrorcheck(hipFree(d_count));

    gpuErrorcheck(hipFree(d_mutex));

    gpuErrorcheck(hipFree(d_output));

    hipDeviceSynchronize();
}

void InitDistribution::update()
{
    float elapsedTime;
    hipEventCreate(&start_global);
    hipEventCreate(&stop_global);
    hipEventRecord(start_global, 0);

    float elapsedTimeKernel;

    elapsedTimeKernel = kernel::resetArrays(d_mutex, d_x, d_y, d_z, d_mass, d_count, d_start, d_sorted, d_child, d_index,
                        d_min_x, d_max_x, d_min_y, d_max_y, d_min_z, d_max_z, numParticles, numNodes);

    std::cout << "\tElapsed time: " << elapsedTimeKernel << std::endl;

    elapsedTimeKernel = kernel::computeBoundingBox(d_mutex, d_x, d_y, d_z, d_min_x, d_max_x, d_min_y, d_max_y,
                               d_min_z, d_max_z, numParticles);

    std::cout << "\tElapsed time: " << elapsedTimeKernel << std::endl;

    elapsedTimeKernel = kernel::buildTree(d_x, d_y, d_z, d_mass, d_count, d_start, d_child, d_index, d_min_x, d_max_x, d_min_y, d_max_y,
                      d_min_z, d_max_z, numParticles, numNodes);

    std::cout << "\tElapsed time: " << elapsedTimeKernel << std::endl;

    elapsedTimeKernel = kernel::centreOfMass(d_x, d_y, d_z, d_mass, d_index, numParticles);

    std::cout << "\tElapsed time: " << elapsedTimeKernel << std::endl;

    elapsedTimeKernel = kernel::sort(d_count, d_start, d_sorted, d_child, d_index, numParticles);

    std::cout << "\tElapsed time: " << elapsedTimeKernel << std::endl;

    elapsedTimeKernel = kernel::computeForces(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_ax, d_ay, d_az, d_mass, d_sorted, d_child,
                          d_min_x, d_max_x, numParticles, parameters.gravity);

    std::cout << "\tElapsed time: " << elapsedTimeKernel << std::endl;

    elapsedTimeKernel = kernel::update(d_x, d_y, d_z, d_vx, d_vy, d_vz, d_ax, d_ay, d_az, numParticles,
                   parameters.timestep, parameters.dampening);

    std::cout << "\tElapsed time: " << elapsedTimeKernel << std::endl;

    //FillOutputArray(d_x, d_y, d_output, numNodes);

    hipEventRecord(stop_global, 0);
    hipEventSynchronize(stop_global);
    hipEventElapsedTime(&elapsedTime, start_global, stop_global);
    hipEventDestroy(start_global);
    hipEventDestroy(stop_global);

    std::cout << "Elapsed time for step " << step << " : " << elapsedTime << std::endl;

    step++;
}


void InitDistribution::plummerModel(float *mass, float *x, float* y, float *z,
                                    float *x_vel, float *y_vel, float *z_vel,
                                    float *x_acc, float *y_acc, float *z_acc, int n)
{
    float a = 1.0;
    float pi = 3.14159265;
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(0, 1.0);
    std::uniform_real_distribution<float> distribution2(0, 0.1);
    std::uniform_real_distribution<float> distribution_phi(0.0, 2 * pi);
    std::uniform_real_distribution<float> distribution_theta(-1.0, 1.0);

    // loop through all particles
    for (int i = 0; i < n; i++){
        float phi = distribution_phi(generator);
        float theta = acos(distribution_theta(generator));
        float r = a / sqrt(pow(distribution(generator), -0.666666) - 1);

        // set mass and position of particle
        mass[i] = 1.0;
        x[i] = r*cos(phi);
        y[i] = r*sin(phi);
        z[i] = 0.0;

        // set velocity of particle
        float s = 0.0;
        float t = 0.1;
        while(t > s*s*pow(1.0 - s*s, 3.5)){
            s = distribution(generator);
            t = distribution2(generator);
        }
        float v = 100*s*sqrt(2)*pow(1.0 + r*r, -0.25);
        phi = distribution_phi(generator);
        theta = acos(distribution_theta(generator));
        x_vel[i] = v*cos(phi);
        y_vel[i] = v*sin(phi);
        z_vel[i] = 0.0;

        // set acceleration to zero
        x_acc[i] = 0.0;
        y_acc[i] = 0.0;
        z_acc[i] = 0.0;
    }
}



