#include "hip/hip_runtime.h"
//
// Created by Michael Staneker on 23.02.21.
//

#include "../include/InitDistribution.h"

void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) getchar();
    }
}

InitDistribution::InitDistribution() {

    step = 0;
    numParticles = NUM_BODIES;
    numNodes = 2 * numParticles + 12000;

    // allocate host data
    h_x_min = new float;
    h_x_max = new float;
    h_y_min = new float;
    h_y_max = new float;
    h_y_min = new float;
    h_y_max = new float;

    h_mass = new float[numNodes];

    h_x = new float[numNodes];
    h_y = new float[numNodes];
    h_z = new float[numNodes];

    h_vx = new float[numNodes];
    h_vy = new float[numNodes];
    h_vz = new float[numNodes];

    h_ax = new float[numNodes];
    h_ay = new float[numNodes];
    h_az = new float[numNodes];

    h_child = new int[8*numNodes];
    
    h_start = new int[numNodes];
    h_sorted = new int[numNodes];
    h_count = new int[numNodes];
    h_output = new float[2*numNodes];

    // allocate device data
    gpuErrorcheck(hipMalloc((void**)&d_min_x, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_x, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_min_y, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_y, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_min_z, sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_max_z, sizeof(float)));

    gpuErrorcheck(hipMemset(d_min_x, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_x, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_min_y, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_y, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_min_z, 0, sizeof(float)));
    gpuErrorcheck(hipMemset(d_max_z, 0, sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_mass, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_x, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_y, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_z, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_vx, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_vy, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_vz, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_ax, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_ay, numNodes*sizeof(float)));
    gpuErrorcheck(hipMalloc((void**)&d_az, numNodes*sizeof(float)));

    gpuErrorcheck(hipMalloc((void**)&d_index, sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_child, 8*numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_start, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_sorted, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_count, numNodes*sizeof(int)));
    gpuErrorcheck(hipMalloc((void**)&d_mutex, sizeof(int)));

    gpuErrorcheck(hipMemset(d_start, -1, numNodes*sizeof(int)));
    gpuErrorcheck(hipMemset(d_sorted, 0, numNodes*sizeof(int)));

    int memSize = sizeof(float) * 2 * numParticles;

    gpuErrorcheck(hipMalloc((void**)&d_output, 2*numNodes*sizeof(float)));

    plummerModel(h_mass, h_x, h_y, h_z, h_vx, h_vy, h_vz, h_ax, h_ay, h_az, numParticles);

    // copy data to GPU device
    hipMemcpy(d_mass, h_mass, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z, h_z, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vx, h_vx, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vy, h_vy, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vz, h_vz, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ax, h_ax, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ay, h_ay, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_az, h_az, 2*numParticles*sizeof(float), hipMemcpyHostToDevice);

}

InitDistribution::~InitDistribution() {
    delete h_min_x;
    delete h_max_x;
    delete h_min_y;
    delete h_max_y;
    delete h_min_z;
    delete h_max_z;

    delete [] h_mass;

    delete [] h_x;
    delete [] h_y;
    delete [] h_z;

    delete [] h_vx;
    delete [] h_vy;
    delete [] h_vz;

    delete [] h_ax;
    delete [] h_ay;
    delete [] h_az;

    delete [] h_child;
    delete [] h_start;
    delete [] h_sorted;
    delete [] h_count;
    delete [] h_output;

    gpuErrorcheck(hipFree(d_min_x));
    gpuErrorcheck(hipFree(d_max_x));
    gpuErrorcheck(hipFree(d_min_y));
    gpuErrorcheck(hipFree(d_max_y));
    gpuErrorcheck(hipFree(d_min_z));
    gpuErrorcheck(hipFree(d_max_z));

    gpuErrorcheck(hipFree(d_mass));

    gpuErrorcheck(hipFree(d_x));
    gpuErrorcheck(hipFree(d_y));
    gpuErrorcheck(hipFree(d_z));

    gpuErrorcheck(hipFree(d_vx));
    gpuErrorcheck(hipFree(d_vy));
    gpuErrorcheck(hipFree(d_vz));

    gpuErrorcheck(hipFree(d_ax));
    gpuErrorcheck(hipFree(d_ay));
    gpuErrorcheck(hipFree(d_az));

    gpuErrorcheck(hipFree(d_index));
    gpuErrorcheck(hipFree(d_child));
    gpuErrorcheck(hipFree(d_start));
    gpuErrorcheck(hipFree(d_sorted));
    gpuErrorcheck(hipFree(d_count));

    gpuErrorcheck(hipFree(d_mutex));

    gpuErrorcheck(hipFree(d_output));

    hipDeviceSynchronize();
}

void InitDistribution::update()
{
    float elapsedTime;
    hipEventCreate(&start_global);
    hipEventCreate(&stop_global);
    hipEventRecord(start_global, 0);

    kernel::resetArrays(d_mutex, d_x, d_y, d_mass, d_count, d_start, d_sorted, d_child, d_index, d_left, d_right, d_bottom, d_top, numParticles, numNodes);
    kernel::computeBoundingBox(d_mutex, d_x, d_y, d_left, d_right, d_bottom, d_top, numParticles);
    kernel::buildTree(d_x, d_y, d_mass, d_count, d_start, d_child, d_index, d_left, d_right, d_bottom, d_top, numParticles, numNodes);
    kernel::centreOfMass(d_x, d_y, d_mass, d_index, numParticles);
    kernel::sort(d_count, d_start, d_sorted, d_child, d_index, numParticles);
    kernel::computeForces(d_x, d_y, d_vx, d_vy, d_ax, d_ay, d_mass, d_sorted, d_child, d_left, d_right, numParticles, parameters.gravity);
    kernel::update(d_x, d_y, d_vx, d_vy, d_ax, d_ay, numParticles, parameters.timestep, parameters.dampening);
    //FillOutputArray(d_x, d_y, d_output, numNodes);

    hipEventRecord(stop_global, 0);
    hipEventSynchronize(stop_global);
    hipEventElapsedTime(&elapsedTime, start_global, stop_global);
    hipEventDestroy(start_global);
    hipEventDestroy(stop_global);

    std::cout << "Elapsed time for step " << step << " : " << elapsedTime << std::endl;

    step++;
}


void InitDistribution::plummerModel(float *mass, float *x, float* y, float *z,
                                    float *x_vel, float *y_vel, float *z_vel,
                                    float *x_acc, float *y_acc, float *z_acc, int n)
{
    float a = 1.0;
    float pi = 3.14159265;
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(0, 1.0);
    std::uniform_real_distribution<float> distribution2(0, 0.1);
    std::uniform_real_distribution<float> distribution_phi(0.0, 2 * pi);
    std::uniform_real_distribution<float> distribution_theta(-1.0, 1.0);

    // loop through all particles
    for (int i = 0; i < n; i++){
        float phi = distribution_phi(generator);
        float theta = acos(distribution_theta(generator));
        float r = a / sqrt(pow(distribution(generator), -0.666666) - 1);

        // set mass and position of particle
        mass[i] = 1.0;
        x[i] = r*cos(phi);
        y[i] = r*sin(phi);
        z[i] = 0.0;

        // set velocity of particle
        float s = 0.0;
        float t = 0.1;
        while(t > s*s*pow(1.0 - s*s, 3.5)){
            s = distribution(generator);
            t = distribution2(generator);
        }
        float v = 100*s*sqrt(2)*pow(1.0 + r*r, -0.25);
        phi = distribution_phi(generator);
        theta = acos(distribution_theta(generator));
        x_vel[i] = v*cos(phi);
        y_vel[i] = v*sin(phi);
        z_vel[i] = 0.0;

        // set acceleration to zero
        x_acc[i] = 0.0;
        y_acc[i] = 0.0;
        z_acc[i] = 0.0;
    }
}



